#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

// code to initialize a vector
void vector_init(float *a, int n)
{
    for(int i = 0 ; i < n ; i++)
    {
        a[i] = (float)(rand() % 100);
    }
}

// verify the result
void verify_result(float *a, float *b, float *c, float factor, int n)
{
    for(int i = 0 ; i < n ; i++)
    {
        assert(c[i] == factor * a[i] + b[i]);
    }
}


int main()
{   // Vector size
    int n = 1<<16;
    size_t bytes = n * sizeof(float);

    // Declare vector pointers
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b;

    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
    
    // Allocate the memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);

    vector_init(h_a, n);
    vector_init(h_b, n);

    // create and initialize a new context
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // copy the vectors to the device
    hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1);
    hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1);

    // Launch a simple Saxpy kernel
    // Functiom signature: handle, # elements n, A, increment, B, increment
    const float scale = 2.0f;
    hipblasSaxpy(handle, n, &scale, d_a, 1, d_b, 1);

    // Copy the result vector back out
    hipblasGetVector(n, sizeof(float), d_b, 1, h_c, 1);

    verify_result(h_a, h_b, h_c, scale, n);

    // clean up the created handle
    hipblasDestroy(handle);

    // Release allocated memory
    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);

    
    return 0;


}