
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>

/*
1-D convolution kernel
Arguments :
    array = padded array
    mask  = convolution array
    result = result array
    n = number of elements in array
    m = number of elements  in  the mask
*/
__global__ void convolution_1d(int *array, int *mask, int* result, int n, int m)
{
    // Global thread ID calculation
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate radius of the mask
    int r = m/2;

    // Calculate the starting point for the element
    int start = tid - r;

    // Temp value for calculation
    int temp = 0;

    // Go over each element of the mask
    for( int j = 0; j < m; j++)
    {
        // Ignore elements that hang off
        if((start + j >= 0) && (start +j < n))
        {
            temp += array[start + j ] * mask[j];
        }
    } 
    
    // Write back the results
    result[tid] = temp;
}

// Verify the result on the CPU
void verify_result(int *array, int *mask, int *result, int n, int m)
{
    int radius = m/2;
    int temp;
    int start;

    for(int i = 0 ; i < n; i++)
    {
        start = i - radius;
        temp = 0;
        for(int j = 0; j < m ; j++)
        {
            if((start + j >= 0) && (start + j < n))
            {
                temp += array[start + j ] * mask[j];
            }
        }
        assert( temp == result[i]);
    }
}

int main()
{   // Number of elements in result array
    int n = 1<<20;

    // Size of array in bytes
    int bytes_n = n * sizeof(int);

    // Number of elements in the convolutional mask
    int m = 7;

    // Size of mask in bytes
    int bytes_m = m * sizeof(int);

    // Allocate the array on host device
    std::vector<int> h_array(n);

    // initialize the array
    std::generate(begin(h_array), end(h_array), [](){ return rand() % 100;});

    // Allocate the mask and initialize it 
    std::vector<int> h_mask(m);
    std::generate(begin(h_mask), end(h_mask), [](){ return rand() % 10;});

    // Allocate space for the result
    std::vector<int> h_result(n);
    
    // Allocate space on the device
    int *d_array, *d_mask, *d_result;
    hipMalloc(&d_array, bytes_n);
    hipMalloc(&d_mask, bytes_m);
    hipMalloc(&d_result, bytes_n);

    // Copy the data to the device
    hipMemcpy(d_array, h_array.data(), bytes_n, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask.data(), bytes_m, hipMemcpyHostToDevice);

    // Threads per Thread Block
    int THREADS = 256;

    // Number of Thread Blocks
    int GRID = (n + THREADS -1)/ THREADS;

    // call the kernel
    convolution_1d<<< GRID, THREADS>>>(d_array, d_mask, d_result, n, m);

    // Copy back the result
    hipMemcpy(h_result.data(), d_result, bytes_n, hipMemcpyDeviceToHost);

    // verify the result
    verify_result(h_array.data(), h_mask.data(), h_result.data(), n, m);

    std::cout <<"COMPLETED SUCCESSFULLY\n";

    // Free allocated memory on the device and the host
    hipFree(d_result);
    hipFree(d_mask);
    hipFree(d_array);

    return 0;

}