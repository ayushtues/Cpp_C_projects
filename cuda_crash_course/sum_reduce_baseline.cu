
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cassert>
#include <numeric>

using std::accumulate;
using std::generate;
using std::cout;
using std::vector;

#define SHMEM_SIZE 256

__global__ void sum_reduction(int *v, int *v_r)
{   
    // Allocate shared memory
    __shared__ int partial_sum[SHMEM_SIZE];

    // Calculate thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Load elements into shared memory
    partial_sum[threadIdx.x] = v[tid];
    __syncthreads();

    // Iterate of logbase 2 the block dimension
    for(int s = 1; s<blockDim.x; s*=2 )
    {
        // Reduce the threads performing work by half the previous iteration each cycle
        if(threadIdx.x % (2*s) == 0 )
        {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x +s];
        }    
        __syncthreads();
    }

    // Let the thread 0 for this block write it's results to main memory
    // Result indexed by this block
    if(threadIdx.x == 0 )
    {
        v_r[blockIdx.x] = partial_sum[0];
    }

}

int main()
{   
    // vector size
    int N = 1<<16;
    size_t bytes = N * sizeof(int);

    // host data
    vector<int> h_v(N);
    vector<int> h_v_r(N);

    // initialize vector
    generate(begin(h_v), end(h_v), [](){return rand() % 10; });

    // device memory
    int *d_v, *d_v_r;

    hipMalloc(&d_v, bytes);
    hipMalloc(&d_v_r, bytes);

    // copy from host ( CPU ) to device ( GPU )
    hipMemcpy(d_v, h_v.data(), bytes, hipMemcpyHostToDevice);
    
    // Thread block size
    const int TB_SIZE = 256;

    // The Grid size
    int GRID_SIZE = N / TB_SIZE;

    // call the kernels
    sum_reduction<<<GRID_SIZE, TB_SIZE>>>(d_v, d_v_r); // first convert the 65536 vector into a 256 sized vector of partial sums
    sum_reduction<<<1, TB_SIZE>>>(d_v_r, d_v_r); // use the 256 sized vector of partial sums to calculate the final sum

    hipMemcpy(h_v_r.data(), d_v_r, bytes, hipMemcpyDeviceToHost);

    // check the result
    assert(h_v_r[0] == std::accumulate(begin(h_v), end(h_v), 0));

    cout<<"COMPLETED SUCCESSFULLY\n";

    return 0;


}